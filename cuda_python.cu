#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include "main.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <string>
#include <boost/python.hpp>
#include <boost/python/numpy.hpp>

//TODO:
//Fix cuda errors when size is large

#define cudaCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void setupKernel(hiprandState *states) {
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(1234, id, 0, &states[id]);  // 	Initialize CURAND
}

__global__ void gpuTick(hiprandState *randStates, const Cell *board, Cell *boardCopy, const Params *params,
                        const unsigned int width, const unsigned int height) {
    const unsigned int size = width * height;
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= size) return;
    hiprandState localRandState = randStates[id];

    unsigned int x = id % width;
    unsigned int y = id / width;

    Cell cell = board[id];
    float newFuel = cell.fuel - cell.fireActivity * params->burnRate;

    const int ns[3] = {-1, 0, 1};
    float activityGrid[8];
    int dirIndex = -1;
    for (int xOffset: ns) {
        for (int yOffset: ns) {
            dirIndex++;
            if (xOffset == 0 && yOffset == 0)
                continue; // don't count same cell as neighbour
            // calculate neighbour coordinate
            int nX = (int) x + xOffset;
            int nY = (int) y + yOffset;
            if (nX >= width || nY >= height || nX < 0 || nY < 0) {
                activityGrid[dirIndex] = cell.fireActivity;
                continue;
            }
            unsigned int nI = nY * width + nX;
            // ------ WIND ------
            // Fire activity from neighbour cell counts more if wind comes from there
            activityGrid[dirIndex] =
                    board[nI].fireActivity * (1 + params->windMatrix[dirIndex] * params->windEffectMultiplier);
            // ------ HEIGHT ------
            // Same but for height, going down decreases activity spread, going up increases it
            float heightDifference = cell.height - board[nI].height;
//             hD > 0 when neighbouring cell is higher than neighbour (fire would spread up)
//             hD < 0 when neighbouring cell is lower than neighbour (fire would spread down)
            heightDifference *= heightDifference > 0 ?
                                params->heightEffectMultiplierUp :
                                params->heightEffectMultiplierDown;
            activityGrid[dirIndex] = activityGrid[dirIndex] * (heightDifference + 1);
        }
    }
    float activitySum = 0;
    for (float activity: activityGrid)
        activitySum += activity;
    float activity = (activitySum / 8) * cell.landCoverSpreadRate;
    float newActivity = cell.fireActivity;
    float randomNum = hiprand_uniform(&localRandState);
//    float randomNum = .5;
    if (activity > params->activityThreshold + randomNum / 5) {
//        // Increase fire activity in current cell
        newActivity = cell.fuel * activity /
                      (params->cellArea / params->spreadSpeed * params->areaEffectMultiplier);
    } else if (activity <= params->fireDeathThreshold) {
//        // Reduce fire activity in current cell
        newActivity /= 1 + (params->deathRate / (params->cellArea * params->areaEffectMultiplier));
    }

    boardCopy[id] = {
            newActivity,
            newFuel,
            cell.height, cell.landCoverSpreadRate
    };
}

class Simulation {
private:
    unsigned int width;
    unsigned int height;
    unsigned int size;
    hiprandState *d_randState = nullptr;
    Cell *board;
    Cell *d_board = nullptr;
    Cell *d_boardCopy = nullptr;
    Params *d_params = nullptr;
    Params *params = (Params *) malloc(sizeof(Params));
    int nThreads;

public:
    Simulation(unsigned int w, unsigned int h, int threads = 1024) {
        nThreads = threads;
        width = w;
        height = h;
        size = w * h;
        board = new Cell[size];

        params->burnRate = .1;
        params->heightEffectMultiplierUp = 2;
        params->heightEffectMultiplierDown = 1;
        params->windEffectMultiplier = 1;
        params->activityThreshold = .2;
        params->spreadSpeed = 1.5;
        params->deathRate = .2;
        params->areaEffectMultiplier = 1;
        params->fireDeathThreshold = .1;
        //           nw     w     sw     s      n     ne     e     se
        float wm[8] = {1, 2, 3, 5, 0, 1, 2, 5};
        for (int i = 0; i < 8; i++)
            params->windMatrix[i] = wm[i];
        params->cellArea = 1;
        initBoard();
        initCuda();
    }

    ~Simulation() {
        freeCuda();
    }

    [[nodiscard]] unsigned int gridDim() const {
        return size / nThreads + 1;
    }

    void tick(bool print = true) {
        // Execute on GPU
        gpuTick<<<gridDim(), nThreads>>>(
                d_randState, d_board, d_boardCopy, d_params,
                width, height
        );
//        cudaCheck( hipPeekAtLastError() );
//        cudaCheck( hipDeviceSynchronize() );

        if (print) {
            // Copy data back to CPU
            cudaCheck(hipMemcpy(board, d_boardCopy, size * sizeof(Cell), hipMemcpyDeviceToHost));
            printBoard();
        }

        std::swap(d_board, d_boardCopy);
    }

    void initBoard() {
        for (int i = 0; i < size; i++)
            board[i] = {
                    1,
                    1,
                    1,
                    1,
            };
    }

    void initCuda() {
        size_t free, total;
        cudaCheck(hipSetDevice(0));
        cudaCheck(hipMemGetInfo(&free, &total));

        printf("Checking GPU MemInfo: free: %zu, total: %zu\n", free, total);

        // Init [size] random generators on GPU for each thread
        cudaCheck(hipMalloc(&d_randState, gridDim() * nThreads * sizeof(hiprandState)));
        setupKernel<<<gridDim(), nThreads>>>(d_randState);

        // allocate gpu buffers for board and copy
        cudaCheck(hipMalloc((void **) &d_board, size * sizeof(Cell)));
        cudaCheck(hipMalloc((void **) &d_boardCopy, size * sizeof(Cell)));
        // copy board from CPU to GPU
        cudaCheck(hipMemcpy(d_board, board, size * sizeof(Cell), hipMemcpyHostToDevice));

        // allocate gpu buffers for params
        cudaCheck(hipMalloc((void **) &d_params, sizeof(Params)));
        // copy params from CPU to GPU
        cudaCheck(hipMemcpy(d_params, &params, sizeof(Params), hipMemcpyHostToDevice));
    }

    void printBoard() {
        for (int j = 0; j < width * height; j++) {
            auto cell = board[j];
            if (cell.fireActivity > .5)
                printf("O ");
            else
                printf("_ ");
            if (j % width == width - 1)
                printf("\n");
        }
    }

    void freeCuda() {
        hipFree(d_board);
        hipFree(d_boardCopy);
        hipFree(d_randState);
    }
};

int findBestThreadCount(int W = 100, int H = 100) {
    using std::chrono::high_resolution_clock;
    using std::chrono::duration_cast;
    using std::chrono::duration;
    using std::chrono::milliseconds;
    duration<double, std::milli> best = std::chrono::system_clock::duration::max();
    int bestN = -1;
    // warm up cuda boy
    auto sim = Simulation(W, H, 32);
    sim.tick(false);

    for (int n = 96; n <= 192; n += 32) {
        auto t1 = high_resolution_clock::now();

        sim = Simulation(W, H, n);
        for (int i = 0; i < 100; i++) {
            sim.tick(false);
        }

        auto t2 = high_resolution_clock::now();
        /* Getting number of milliseconds as a double. */
        duration<double, std::milli> dur = t2 - t1;
        std::cout << dur.count() << "ms - " << n << " threads" << std::endl;
        if (dur < best) {
            best = dur;
            bestN = n;
        }
    }
    printf("Best thread count is %i with duration %fms\n", bestN, best.count());
    return bestN;
}

int main() {
    findBestThreadCount(50, 50);

    return 0;
}

int batchSimulate(short *landCoverGrid,
                  short *elevation,
                  bool **fire,
                  double ***weather,
                  double *psoConfigs,
                  double *landCoverRates,
                  int width, int height, int timeSteps, int checkpoints,
                  int weatherElements, int psoParams, int landCoverTypes, int batchSize,
                  double output[]) {
    for (int b = 0; b < batchSize; b++) {

    }
    return 12;
}

namespace p = boost::python;
namespace np = boost::python::numpy;


np::ndarray wrapBatchSimulate(np::ndarray const &npLandCoverGrid,
                              np::ndarray const &npElevation,
                              np::ndarray const &npFire,
                              np::ndarray const &npWeather,
                              np::ndarray const &npPsoConfigs,
                              np::ndarray const &npLandCoverRates) {
    // Make sure we get right types
    // 2D WxH array, each cell value is index for landCoverRates array
    if (npLandCoverGrid.get_dtype() != np::dtype::get_builtin<short>()) {
        PyErr_SetString(PyExc_TypeError, "Incorrect landCoverGrid data type");
        p::throw_error_already_set();
    }
    // 2D WxH array
    if (npElevation.get_dtype() != np::dtype::get_builtin<short>()) {
        PyErr_SetString(PyExc_TypeError, "Incorrect elevation data type");
        p::throw_error_already_set();
    }
    // 3D WxHxC array C is fire checkpoint steps count
    if (npFire.get_dtype() != np::dtype::get_builtin<bool>()) {
        PyErr_SetString(PyExc_TypeError, "Incorrect fire data type");
        p::throw_error_already_set();
    }
    // 4D WxHxTxE array T is time steps. E is elements size (wind X, wind Y)
    if (npWeather.get_dtype() != np::dtype::get_builtin<double>()) {
        PyErr_SetString(PyExc_TypeError, "Incorrect weather data type");
        p::throw_error_already_set();
    }

    // 2D PxN array P is params count, N is batch size
    if (npPsoConfigs.get_dtype() != np::dtype::get_builtin<double>()) {
        PyErr_SetString(PyExc_TypeError, "Incorrect psoConfigs data type");
        p::throw_error_already_set();
    }
    // 2D LxN array L is land cover type count, N is batch size
    if (npLandCoverRates.get_dtype() != np::dtype::get_builtin<double>()) {
        PyErr_SetString(PyExc_TypeError, "Incorrect landCoverRates data type");
        p::throw_error_already_set();
    }

    int width = (int) npLandCoverGrid.shape(0);
    auto height = (int) npLandCoverGrid.shape(1);
    auto timeSteps = (int) npWeather.shape(2);
    auto checkpoints = (int) npFire.shape(2);
    auto weatherElements = (int) npWeather.shape(3);
    auto psoParams = (int) npPsoConfigs.shape(0);
    auto batchSize = (int) npPsoConfigs.shape(1);
    auto landCoverTypes = (int) npLandCoverRates.shape(0);

    auto landCoverGrid = reinterpret_cast<short *>(npLandCoverGrid.get_data());
    auto elevation = reinterpret_cast<short *>(npElevation.get_data());
    auto fire = reinterpret_cast<bool **>(npFire.get_data());
    auto weather = reinterpret_cast<double ***>(npWeather.get_data());
    auto psoConfigs = reinterpret_cast<double *>(npPsoConfigs.get_data());
    auto landCoverRates = reinterpret_cast<double *>(npLandCoverRates.get_data());

    static double output[2];

    auto temp = batchSimulate(landCoverGrid, elevation, fire, weather, psoConfigs, landCoverRates,
                              width, height, timeSteps, checkpoints,
                              weatherElements, psoParams, landCoverTypes, batchSize, output);

    output[0] = temp;
    output[1] = temp * 12;
    np::dtype dt = np::dtype::get_builtin<double>();
    p::tuple shape = p::make_tuple(3); // It has shape (2,)
    p::tuple stride = p::make_tuple(sizeof(double)); // 1D array, so its just size of double
    np::ndarray result = np::from_data(output, dt, shape, stride, p::object());
    return result;
}

BOOST_PYTHON_MODULE (cuda_python) {  // Thing in brackets should match output library name
    Py_Initialize();
    np::initialize();
    p::def("batch_simulate", wrapBatchSimulate);
    p::def("find_best_thread_count", findBestThreadCount);
}