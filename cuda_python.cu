#include "hip/hip_runtime.h"
#include "main.h"
#include "pythonHelpers.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <string>

//TODO:
// Remove malloc on repeated Simulations
// add weather

#define cudaCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPU assert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void setupKernel(hiprandState *states, int seed) {
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, id, 0, &states[id]);  // 	Initialize CURAND
}

__global__ void gpuTick(hiprandState *randStates,
                        Cell *board, Cell *boardCopy,
                        const double *landCoverRates,
                        const double *params,
                        const double *weather,
                        const NDimArrayShape lcrShape,
                        const NDimArrayShape paramsShape,
                        const NDimArrayShape weatherShape,
                        const unsigned int batchIndex
) {
    const auto width = weatherShape.s0;
    const auto height = weatherShape.s1;
    const auto size = width * height;
    auto id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= size) return;
//    auto localRandState = randStates[id];

    auto x = id % width;
    auto y = id / width;

    Cell cell = board[id];
    auto newFuel = cell.fuel - cell.fireActivity * params[Params::burnRate + batchIndex * paramsShape.s0];

    const int ns[3] = {-1, 0, 1};
    double activityGrid[8];
    auto dirIndex = -1;
    // weather[x, y, t, e] t = batch index, e = weather element
    // e: 0 -> wind U component (horizontal towards east, +x)
    // e: 1 -> wind V component (vertical towards north, -y)
    auto windUElement = 0;
    auto windVElement = 1;
    auto windX = weather[
            windUElement * weatherShape.s0 * weatherShape.s1 * weatherShape.s2 +
            batchIndex * weatherShape.s0 * weatherShape.s1 +
            y * weatherShape.s0 +
            x
    ];
    auto windY = -1 * weather[
            windVElement * weatherShape.s0 * weatherShape.s1 * weatherShape.s2 +
            batchIndex * weatherShape.s0 * weatherShape.s1 +
            y * weatherShape.s0 +
            x
    ];
    for (auto xOffset: ns) {
        for (auto yOffset: ns) {
            dirIndex++;
            if (xOffset == 0 && yOffset == 0)
                continue; // skip current cell because it's not a neighbour
            // calculate neighbour coordinate
            auto nX = (int) x + xOffset;
            auto nY = (int) y + yOffset;
            if (nX >= width || nY >= height || nX < 0 || nY < 0) {
                activityGrid[dirIndex] = cell.fireActivity;
                continue;
            }
            auto nI = nY * width + nX;
            // ------ WIND ------
            // Fire activity from neighbour cell counts more if wind comes from there
            activityGrid[dirIndex] = board[nI].fireActivity;
            auto wx = windX * xOffset * -1;
            auto wy = windY * yOffset * -1;
            // try to keep windFromNeighbour between 0 and 1
            auto windFromNeighbour =
                    (wx + wy) / 200 * params[Params::windEffectMultiplier + batchIndex * paramCount];
            activityGrid[dirIndex] *= 1 + windFromNeighbour;
            // ------ HEIGHT ------
            // Same but for height, going down decreases activity spread, going up increases it
            double heightDifference = (double) (cell.height - board[nI].height) / 20;
//             hD > 0 when neighbouring cell is higher than neighbour (fire would spread up)
//             hD < 0 when neighbouring cell is lower than neighbour (fire would spread down)
            heightDifference *= heightDifference > 0 ?
                                params[Params::heightEffectMultiplierUp + batchIndex * paramCount] :
                                params[Params::heightEffectMultiplierDown + batchIndex * paramCount];
            activityGrid[dirIndex] *= 1 + heightDifference;
        }
    }
    double activitySum = 0;
    for (auto activity: activityGrid)
        activitySum += activity;
    auto activity = (activitySum / 8) *
                    landCoverRates[cell.landCoverSpreadIndex + batchIndex * lcrShape.s0];
    auto newActivity = cell.fireActivity;
    auto randomNum = hiprand_uniform(randStates + id);
    auto cellArea = params[Params::cellArea + batchIndex * paramsShape.s0];
    auto activityThreshold = params[Params::activityThreshold + batchIndex * paramsShape.s0];
    auto areaEffectMultiplier = params[Params::areaEffectMultiplier + batchIndex * paramsShape.s0];
    auto fireDeathThreshold = params[Params::fireDeathThreshold + batchIndex * paramsShape.s0];
    if (activity > activityThreshold + randomNum / 5) {
        auto spreadSpeed = params[Params::spreadSpeed + batchIndex * paramsShape.s0];
//        // Increase fire activity in current cell
        newActivity = cell.fuel * activity /
                      (cellArea / spreadSpeed * areaEffectMultiplier);
    } else if (activity <= fireDeathThreshold) {
        auto deathRate = params[Params::deathRate + batchIndex * paramsShape.s0];
//        // Reduce fire activity in current cell
        newActivity /= 1 + (deathRate / (cellArea * areaEffectMultiplier));
    }

    boardCopy[id].fireActivity = newActivity;
    boardCopy[id].fuel = newFuel;
    boardCopy[id].height = cell.height;
    boardCopy[id].landCoverSpreadIndex = cell.landCoverSpreadIndex;
}

class Simulation {
private:
    unsigned int width;
    unsigned int height;
    unsigned int size;
    unsigned int batchIndex;
    Cell *board;
    NDimArray<short> landCoverGrid{};
    NDimArray<short> elevation{};
    NDimArray<bool> fire{};
    NDimArray<double> weather{};
    NDimArray<double> psoConfigs{};
    NDimArray<double> landCoverRates{};

    NDimArrayShape lcrShape{};
    NDimArrayShape paramsShape{};
    NDimArrayShape weatherShape{};

    hiprandState *d_randState = nullptr;
    Cell *d_board = nullptr;
    Cell *d_boardCopy = nullptr;
    double *d_weather = nullptr;
    double *d_landCoverRates = nullptr;
    double *d_params = nullptr;
    int nThreads;

public:
    Simulation(unsigned int w, unsigned int h, unsigned int batchIndex, int threads,
               const NDimArray<short> &landCoverGrid,
               const NDimArray<short> &elevation,
               const NDimArray<bool> &fire,
               const NDimArray<double> &weather,
               const NDimArray<double> &params,
               const NDimArray<double> &landCoverRates) {
        this->landCoverGrid = landCoverGrid;
        this->elevation = elevation;
        this->fire = fire;
        this->weather = weather;
        this->psoConfigs = params;
        this->landCoverRates = landCoverRates;
        this->lcrShape.s0 = landCoverGrid.shape[0];
        this->lcrShape.s1 = landCoverGrid.shape[1];
        this->paramsShape.s0 = params.shape[0];
        this->paramsShape.s1 = params.shape[1];
        this->weatherShape.s0 = weather.shape[0];
        this->weatherShape.s1 = weather.shape[1];
        this->weatherShape.s2 = weather.shape[2];
        this->weatherShape.s3 = weather.shape[3];

        width = w;
        height = h;
        size = w * h;
        nThreads = threads;
        this->batchIndex = batchIndex;
        board = new Cell[size];

        initBoard();
        initCuda();
    }

    [[nodiscard]] unsigned int gridDim() const {
        return size / nThreads + 1;
    }

    void tick(bool print = true) {
        // Execute on GPU
        gpuTick<<<gridDim(), nThreads>>>(
                d_randState, d_board, d_boardCopy,
                d_landCoverRates, d_params, d_weather,
                lcrShape, paramsShape, weatherShape,
                batchIndex
        );

        if (print) {
            // Copy data back to CPU
            cudaCheck(hipMemcpy(board, d_boardCopy, size * sizeof(Cell), hipMemcpyDeviceToHost))
            printBoard();
        }

        std::swap(d_board, d_boardCopy);
    }

    void initBoard() {
        for (int y = 0; y < height; y++) {
            for (int x = 0; x < width; x++) {
                auto index = y * width + x;
                board[index] = {
                        fire.array[index] ? 1. : 0.,
                        1,
                        elevation.array[index],
                        landCoverGrid.array[index],
                };
            }
        }
    }

    void initCuda() {
        cudaCheck(hipSetDevice(0))

//        size_t free, total;
//        cudaCheck(hipMemGetInfo(&free, &total));
//        printf("Checking GPU MemInfo: free: %zu, total: %zu\n", free, total);

        // Init [size] random generators on GPU for each thread
        cudaCheck(hipMalloc(&d_randState, gridDim() * nThreads * sizeof(hiprandState)))
        setupKernel<<<gridDim(), nThreads>>>(d_randState, rand()); // NOLINT(cert-msc50-cpp)

        auto weatherSize = nDimArrayLength(weather) * sizeof(double);
        auto psoConfigsSize = nDimArrayLength(psoConfigs) * sizeof(double);
        auto landCoverRatesSize = nDimArrayLength(landCoverRates) * sizeof(double);

        printf("nDimArrayLength(weather) = %lu\n", nDimArrayLength(weather));

        // allocate gpu buffers for board and copy
        cudaCheck(hipMalloc((void **) &d_board, size * sizeof(Cell)))
        cudaCheck(hipMalloc((void **) &d_boardCopy, size * sizeof(Cell)))
        cudaCheck(hipMalloc((void **) &d_weather, weatherSize))
        cudaCheck(hipMalloc((void **) &d_params, psoConfigsSize))
        cudaCheck(hipMalloc((void **) &d_landCoverRates, landCoverRatesSize))
        // copy board from CPU to GPU
        cudaCheck(hipMemcpy(d_board, board, size * sizeof(Cell), hipMemcpyHostToDevice))
        cudaCheck(hipMemcpy(d_weather, weather.array, weatherSize, hipMemcpyHostToDevice))
        cudaCheck(hipMemcpy(d_params, psoConfigs.array, psoConfigsSize, hipMemcpyHostToDevice))
        cudaCheck(hipMemcpy(d_landCoverRates, landCoverRates.array, landCoverRatesSize, hipMemcpyHostToDevice))
    }

    void printBoard() {
        for (int j = 0; j < width * height; j++) {
            auto cell = board[j];
            if (cell.fireActivity <= 0)
                printf(". ");
            else if (cell.fireActivity <= .3)
                printf("c ");
            else if (cell.fireActivity <= .6)
                printf("o ");
            else
                printf("O ");
            if (j % width == width - 1)
                printf("\n");
        }
    }

    static void freeCuda() {
        printf("Free CUDA\n");
        cudaCheck(hipDeviceSynchronize())
        cudaCheck(hipDeviceReset())
    }
};

int batchSimulate(NDimArray<short> landCoverGrid,
                  NDimArray<short> elevation,
                  NDimArray<bool> fire,
                  NDimArray<double> weather,
                  NDimArray<double> psoConfigs,
                  NDimArray<double> landCoverRates,
                  double output[]) {
    auto width = landCoverGrid.shape[0];
    auto height = landCoverGrid.shape[1];
    auto batchSize = psoConfigs.shape[1];
    auto timeSteps = weather.shape[2];
    for (int i = 0; i < batchSize; i++) {
        printf("Iteration %i\n", i);
        auto sim = Simulation(width, height, i, 96,
                              landCoverGrid, elevation, fire, weather, psoConfigs, landCoverRates);
        sim.printBoard();
        for (int t = 0; t < timeSteps; t++) {
            printf("Tick %i\n", t);
            sim.tick(true);
        }
        Simulation::freeCuda();
    }

    return 0;
}

int main() {
    int width = 10;
    int height = 8;
    int timeSteps = 20;
    int checkpoints = 3;
    int weatherElements = 2;
    int psoParams = 10;
    int batchSize = 1;
    int landCoverTypes = 8;

    auto landCoverGrid = createNDimArray<short>(2, new long[2]{width, height}, 1);
    auto landCoverRates = createNDimArray<double>(2, new long[2]{width, height}, 1);
    auto elevation = createNDimArray<short>(2, new long[2]{landCoverTypes, batchSize}, 3);
    auto fire = createNDimArray<bool>(3, new long[3]{width, height, checkpoints}, false);
    auto weather = createNDimArray<double>(4, new long[4]{width, height, timeSteps, weatherElements}, 20);
    auto params = createNDimArray<double>(2, new long[2]{psoParams, batchSize}, 1);

    fire.array[height / 2 * width + width / 2] = true;
    fire.array[(height / 2 + 1) * width + width / 2] = true;
    fire.array[height / 2 * width + width / 2 + 1] = true;
    fire.array[(height / 2 + 1) * width + width / 2 + 1] = true;
//    fire.array[1] = true;
//    fire.array[1 * width + 1] = true;
//    fire.array[1 * width + 0] = true;

    for (int x = 0; x < weather.shape[0]; x++) {
        for (int y = 0; y < weather.shape[1]; y++) {
            for (int z = 0; z < weather.shape[2]; z++) {
                weather.array[1 * weather.shape[0] * weather.shape[1] * weather.shape[2] +
                              z * weather.shape[1] * weather.shape[0] +
                              y * weather.shape[0] +
                              x] = -0;
            }
        }
    }

    params.array[Params::activityThreshold] = 0.2;
    params.array[Params::burnRate] = 0.1;
    params.array[Params::fireDeathThreshold] = 0.1;
    params.array[Params::deathRate] = 0.2;
    params.array[Params::areaEffectMultiplier] = 1;
    params.array[Params::heightEffectMultiplierDown] = 1;
    params.array[Params::heightEffectMultiplierUp] = 1;
    params.array[Params::spreadSpeed] = 1.5;
    params.array[Params::windEffectMultiplier] = 3;

//    printNDimArray(weather, "Weather");

    static double output[2];
    auto temp = batchSimulate(landCoverGrid, elevation, fire, weather, params, landCoverRates, output);
    printf("bs output %i\n", temp);
    printf("output again %f %f\n", output[0], output[1]);

    return 0;
}

np::ndarray wrapBatchSimulate(np::ndarray const &npLandCoverGrid,
                              np::ndarray const &npElevation,
                              np::ndarray const &npFire,
                              np::ndarray const &npWeather,
                              np::ndarray const &npPsoConfigs,
                              np::ndarray const &npLandCoverRates) {
    // Make sure we get right types
    // 2D WxH array, each cell value is index for landCoverRates array
    // 2D WxH array
    // 3D WxHxC array C is fire checkpoint steps count
    // 4D WxHxTxE array T is time steps. E is elements size (wind X, wind Y)
    // 2D PxN array P is params count, N is batch size
    // 2D LxN array L is land cover type count, N is batch size

    printf("STARTING C++ ENGINES\n");

    auto landCoverGrid = npToArray<short>(npLandCoverGrid);
    auto elevation = npToArray<short>(npElevation);
    auto fire = npToArray<bool>(npFire);
    auto weather = npToArray<double>(npWeather);
    auto psoConfigs = npToArray<double>(npPsoConfigs);
    auto landCoverRates = npToArray<double>(npLandCoverRates);

    printf("Sizeof landCoverGrid = %lu\n", sizeOfNDimArray(landCoverGrid));
    printf("Sizeof elevation = %lu\n", sizeOfNDimArray(elevation));
    printf("Sizeof fire = %lu\n", sizeOfNDimArray(fire));
    printf("Sizeof weather = %lu\n", sizeOfNDimArray(weather));
    printf("Sizeof psoConfigs = %lu\n", sizeOfNDimArray(psoConfigs));
    printf("Sizeof landCoverRates = %lu\n", sizeOfNDimArray(landCoverRates));

    static double output[2];

    auto temp = batchSimulate(landCoverGrid, elevation, fire, weather, psoConfigs, landCoverRates, output);
//    auto temp = psoConfigs.array[0];
//    auto temp = weather.array[0];

    output[0] = temp;
    output[1] = temp * 12;
    np::dtype dt = np::dtype::get_builtin<double>();
    p::tuple shape = p::make_tuple(3); // It has shape (2,)
    p::tuple stride = p::make_tuple(sizeof(double)); // 1D array, so its just size of double
    np::ndarray result = np::from_data(output, dt, shape, stride, p::object());
    printf("FINITO\n");
    return result;
}

BOOST_PYTHON_MODULE (cuda_python) {  // Thing in brackets should match output library name
    Py_Initialize();
    np::initialize();
    p::def("batch_simulate", wrapBatchSimulate);
//    p::def("find_best_thread_count", findBestThreadCount);
}