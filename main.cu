#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include "main.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void setup_kernel(hiprandState *state) {
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    hiprand_init(1234, idx, 0, &state[idx]);
}

__global__ void gpuTick(hiprandState *randState, const Cell *board, Cell *boardCopy, const Params params,
                        const unsigned int width, const unsigned int height) {
    const unsigned int size = width * height;
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= size) return;

    unsigned int x = id % width;
    unsigned int y = id / width;

    Cell cell = board[id];
    float newFuel = cell.fuel - cell.fireActivity * params.burnRate;

    const int ns[3] = {-1, 0, 1};
    float activityGrid[8];
    int dirIndex = -1;
    for (int xOffset: ns) {
        for (int yOffset: ns) {
            dirIndex++;
            if (xOffset == 0 && yOffset == 0)
                continue; // don't count same cell as neighbour
            // calculate neighbour coordinate
            int nX = (int) x + xOffset;
            int nY = (int) y + yOffset;
            if (nX >= width || nY >= height || nX < 0 || nY < 0) {
                activityGrid[dirIndex] = cell.fireActivity;
                continue;
            }
            unsigned int nI = nY * width + nX;
            // ------ WIND ------
            // Fire activity from neighbour cell counts more if wind comes from there
            activityGrid[dirIndex] = board[nI].fireActivity * params.windMatrix[dirIndex];
            // ------ HEIGHT ------
            // Same but for height, going down decreases activity spread, going up increases it
            float heightDifference = cell.height - board[nI].height;
            // hD > 0 when neighbouring cell is higher than neighbour (fire would spread up)
            // hd < 0 when neighbouring cell is lower than neighbour (fire would spread down)
            heightDifference *= heightDifference > 0 ?
                                params.heightEffectMultiplierUp :
                                params.heightEffectMultiplierDown;
            activityGrid[dirIndex] = activityGrid[dirIndex] * heightDifference + 1;
        }
    }
    float activitySum = 0;
    for (float activity: activityGrid)
        activitySum += activity;
    float activity = (activitySum / 8) * cell.landCoverSpreadRate;
    float newActivity = cell.fireActivity;
    if (activity > params.activityThreshold + hiprand_uniform(randState + id) / 5) {
        // Increase fire activity in current cell
        newActivity = cell.fuel * activity /
                      (params.cellArea / params.spreadSpeed * params.areaEffectMultiplier);
    } else if (activity <= params.fireDeathThreshold) {
        // Reduce fire activity in current cell
        newActivity /= 1 + (params.deathRate / (params.cellArea * params.areaEffectMultiplier));
    }

    boardCopy[id] = {
            newActivity,
            newFuel,
            cell.height, cell.landCoverSpreadRate
    };
}

class Simulation {
private:
    unsigned int width;
    unsigned int height;
    unsigned int size;
    hiprandState *d_randState = nullptr;
    bool cudaFailed = false;
    Cell *board;
    Cell *d_board = nullptr;
    Cell *d_boardCopy = nullptr;
    Params params{};
    hipError_t cudaStatus = hipError_t();
    int nThreads;

public:
    Simulation(unsigned int w, unsigned int h, int threads = 1024) {
        nThreads = threads;
        width = w;
        height = h;
        size = w * h;
        board = new Cell[size];
        params = {
                .1,
                2,
                1,
                .2,
                1.5,
                .2,
                1,
                .1,
                //           nw     w     sw     s      n     ne     e     se
                {1, 2, 3, 5, 0, 1, 2, 5},
        };
        initBoard();
        if (!initCuda()) {
            return;
        }
    }

    ~Simulation() {
        freeCuda();
    }

    bool tick(bool print = true) {
        if (cudaFailed)return false;
        // Execute on GPU
        gpuTick<<<size / nThreads + 1, nThreads>>>(
                d_randState, d_board, d_boardCopy,
                params, width, height
        );

        if (print) {
            // Copy data back to CPU
            cudaStatus = hipMemcpy(board, d_boardCopy, size * sizeof(Cell), hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess)
                return handleError("memCpy to CPU");
            printBoard();
        }

        std::swap(d_board, d_boardCopy);
        return true;
    }

    void initBoard() {
        for (int i = 0; i < size; i++)
            board[i] = {
                    1,
                    1,
                    1,
                    1,
            };
    }

    bool initCuda() {
        // Init random generator on GPU
        cudaStatus = hipMalloc(&d_randState, sizeof(hiprandState));
        if (cudaStatus != hipSuccess)
            return handleError("malloc random");
        setup_kernel<<<size / nThreads + 1, nThreads>>>(d_randState);

        //
        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess)
            return handleError("set device");

        // allocate gpu buffers for board and copy
        cudaStatus = hipMalloc((void **) &d_board, size * sizeof(Cell));
        if (cudaStatus != hipSuccess)
            return handleError("malloc d_board");

        cudaStatus = hipMalloc((void **) &d_boardCopy, size * sizeof(Cell));
        if (cudaStatus != hipSuccess)
            return handleError("malloc d_boardCopy");

        // copy board from CPU to GPU
        cudaStatus = hipMemcpy(d_board, board, size * sizeof(Cell), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess)
            return handleError("memCpy to GPU");

        return true;
    }

    void printBoard() {
        if (cudaFailed)return;
        for (int j = 0; j < width * height; j++) {
            auto cell = board[j];
            if (cell.fireActivity > .5)
                printf("O ");
            else
                printf("_ ");
            if (j % width == width - 1)
                printf("\n");
        }
    }

    void freeCuda() {
        hipFree(d_board);
        hipFree(d_boardCopy);
        hipFree(d_randState);
    }

    bool handleError(const std::string &reason) {
        cudaFailed = true;
        freeCuda();
        printf("Cuda error! %s\n", reason.c_str());
        return false;
    }
};

const int W = 100;
const int H = 100;

int findBestThreadCount() {
    using std::chrono::high_resolution_clock;
    using std::chrono::duration_cast;
    using std::chrono::duration;
    using std::chrono::milliseconds;
    duration<double, std::milli> best = std::chrono::system_clock::duration::max();
    int bestN = -1;
    // warm up cuda boy
    auto sim = Simulation(W, H, 1);
    sim.tick(false);

    for (int n = 32; n <= 1024; n += 32) {
        auto t1 = high_resolution_clock::now();

        sim = Simulation(W, H, n);
        for (int i = 0; i < 1000; i++) {
            sim.tick(false);
        }

        auto t2 = high_resolution_clock::now();
        /* Getting number of milliseconds as a double. */
        duration<double, std::milli> dur = t2 - t1;
        std::cout << dur.count() << "ms - " << n << " threads" << std::endl;
        if (dur < best) {
            best = dur;
            bestN = n;
        }
    }
    printf("Best thread count is %i with duration %fms\n", bestN, best.count());
    return bestN;
}

int main() {
    int nThreads = findBestThreadCount();

//    auto sim = Simulation(W, H, nThreads);
//    for (int i = 0; i < 50; i++) {
//        printf("----------- ITERATION %i -----------\n", i + 1);
//        sim.tick(true);
//    }
    return 0;
}
